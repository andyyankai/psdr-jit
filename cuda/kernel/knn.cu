#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cuda/host/knn.cuh>

#define BLOCK_DIM 16

__global__ void compute_distances(float * ref,
                                  int     ref_width,
                                  int     ref_pitch,
                                  float * query,
                                  int     query_width,
                                  int     query_pitch,
                                  int     height,
                                  float * dist) {

    // Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
    __shared__ int begin_A;
    __shared__ int begin_B;
    __shared__ int step_A;
    __shared__ int step_B;
    __shared__ int end_A;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Initializarion of the SSD for the current thread
    float ssd = 0.f;

    // Loop parameters
    begin_A = BLOCK_DIM * blockIdx.y;
    begin_B = BLOCK_DIM * blockIdx.x;
    step_A  = BLOCK_DIM * ref_pitch;
    step_B  = BLOCK_DIM * query_pitch;
    end_A   = begin_A + (height-1) * ref_pitch;

    // Conditions
    int cond0 = (begin_A + tx < ref_width); // used to write in shared memory
    int cond1 = (begin_B + tx < query_width); // used to write in shared memory & to computations and to write in output array 
    int cond2 = (begin_A + ty < ref_width); // used to computations and to write in output matrix

    // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
    for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {

        // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
        if (a/ref_pitch + ty < height) {
            shared_A[ty][tx] = (cond0)? ref[a + ref_pitch * ty + tx] : 0;
            shared_B[ty][tx] = (cond1)? query[b + query_pitch * ty + tx] : 0;
        }
        else {
            shared_A[ty][tx] = 0;
            shared_B[ty][tx] = 0;
        }

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
        if (cond2 && cond1) {
            for (int k = 0; k < BLOCK_DIM; ++k){
                float tmp = shared_A[k][ty] - shared_B[k][tx];
                ssd += tmp*tmp;
            }
        }

        // Synchronize to make sure that the preceeding computation is done before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory; each thread writes one element
    if (cond2 && cond1) {
        dist[ (begin_A + ty) * query_pitch + begin_B + tx ] = ssd;
    }
}

__global__ void compute_distance_texture(hipTextureObject_t ref,
                                         int                 ref_width,
                                         float *             query,
                                         int                 query_width,
                                         int                 query_pitch,
                                         int                 height,
                                         float*              dist) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if ( xIndex<query_width && yIndex<ref_width) {
        float ssd = 0.f;
        for (int i=0; i<height; i++) {
            float tmp  = tex2D<float>(ref, (float)yIndex, (float)i) - query[i * query_pitch + xIndex];
            ssd += tmp * tmp;
        }
        dist[yIndex * query_pitch + xIndex] = ssd;
    }
}

__global__ void modified_insertion_sort(float * dist,
                                        int     dist_pitch,
                                        int *   index,
                                        int     index_pitch,
                                        int     width,
                                        int     height,
                                        int     k){

    // Column position
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Do nothing if we are out of bounds
    if (xIndex < width) {

        // Pointer shift
        float * p_dist  = dist  + xIndex;
        int *   p_index = index + xIndex;

        // Initialise the first index
        p_index[0] = 0;

        // Go through all points
        for (int i=1; i<height; ++i) {

            // Store current distance and associated index
            float curr_dist = p_dist[i*dist_pitch];
            int   curr_index  = i;

            // Skip the current value if its index is >= k and if it's higher the k-th slready sorted mallest value
            if (i >= k && curr_dist >= p_dist[(k-1)*dist_pitch]) {
                continue;
            }

            // Shift values (and indexes) higher that the current distance to the right
            int j = min(i, k-1);
            while (j > 0 && p_dist[(j-1)*dist_pitch] > curr_dist) {
                p_dist[j*dist_pitch]   = p_dist[(j-1)*dist_pitch];
                p_index[j*index_pitch] = p_index[(j-1)*index_pitch];
                --j;
            }

            // Write the current distance and index at their position
            p_dist[j*dist_pitch]   = curr_dist;
            p_index[j*index_pitch] = curr_index; 
        }
    }
}

__global__ void compute_sqrt(float * dist, int width, int pitch, int k){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex]);
}

__global__ void compute_squared_norm(float * array, int width, int pitch, int height, float * norm){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (xIndex<width){
        float sum = 0.f;
        for (int i=0; i<height; i++){
            float val = array[i*pitch+xIndex];
            sum += val*val;
        }
        norm[xIndex] = sum;
    }
}

__global__ void add_reference_points_norm(float * array, int width, int pitch, int height, float * norm){
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int xIndex = blockIdx.x * blockDim.x + tx;
    unsigned int yIndex = blockIdx.y * blockDim.y + ty;
    __shared__ float shared_vec[16];
    if (tx==0 && yIndex<height)
        shared_vec[ty] = norm[yIndex];
    __syncthreads();
    if (xIndex<width && yIndex<height)
        array[yIndex*pitch+xIndex] += shared_vec[ty];
}

__global__ void add_query_points_norm_and_sqrt(float * array, int width, int pitch, int k, float * norm){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        array[yIndex*pitch + xIndex] = sqrt(array[yIndex*pitch + xIndex] + norm[xIndex]);
}

bool psdr_cuda::knn_cuda_global(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,
                     int *         knn_index) {
    // Constants
    const unsigned int size_of_float = sizeof(float);
    const unsigned int size_of_int   = sizeof(int);

    // Return variables
    hipError_t err0, err1, err2, err3;

    // Check that we have at least one CUDA device 
    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    // Allocate global memory
    float * ref_dev   = NULL;
    float * query_dev = NULL;
    float * dist_dev  = NULL;
    int   * index_dev = NULL;
    size_t  ref_pitch_in_bytes;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&ref_dev,   &ref_pitch_in_bytes,   ref_nb   * size_of_float, dim);
    err1 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err2 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err3 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess) {
        printf("ERROR: Memory allocation error\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Deduce pitch values
    size_t ref_pitch   = ref_pitch_in_bytes   / size_of_float;
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;

    // Check pitch values
    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Copy reference and query data from the host to the device
    err0 = hipMemcpy2D(ref_dev,   ref_pitch_in_bytes,   ref,   ref_nb * size_of_float,   ref_nb * size_of_float,   dim, hipMemcpyHostToDevice);
    err1 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Compute the squared Euclidean distances
    dim3 block0(BLOCK_DIM, BLOCK_DIM, 1);
    dim3 grid0(query_nb / BLOCK_DIM, ref_nb / BLOCK_DIM, 1);
    if (query_nb % BLOCK_DIM != 0) grid0.x += 1;
    if (ref_nb   % BLOCK_DIM != 0) grid0.y += 1;

    compute_distances<<<grid0, block0>>>(ref_dev, ref_nb, ref_pitch, query_dev, query_nb, query_pitch, dim, dist_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Compute the square root of the k smallest distances
    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (k % 16 != 0)        grid2.y += 1;
    compute_sqrt<<<grid2, block2>>>(dist_dev, query_nb, query_pitch, k);    
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Copy k smallest distances / indexes from the device to the host
    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Memory clean-up
    hipFree(ref_dev);
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev); 

    return true;
}

bool psdr_cuda::knn_cuda_texture(const float * ref,
                      int           ref_nb,
                      const float * query,
                      int           query_nb,
                      int           dim,
                      int           k,
                      float *       knn_dist,
                      int *         knn_index) {

    // Constants
    unsigned int size_of_float = sizeof(float);
    unsigned int size_of_int   = sizeof(int);   

    // Return variables
    hipError_t err0, err1, err2;

    // Check that we have at least one CUDA device 
    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    // Allocate global memory
    float * query_dev = NULL;
    float * dist_dev  = NULL;
    int *   index_dev = NULL;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err1 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err2 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess) {
        printf("ERROR: Memory allocation error (hipMallocPitch)\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Deduce pitch values
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;

    // Check pitch values
    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Copy query data from the host to the device
    err0 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);        
        return false; 
    }

    // Allocate CUDA array for reference points
    hipArray* ref_array_dev = NULL;
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    err0 = hipMallocArray(&ref_array_dev, &channel_desc, ref_nb, dim);
    if (err0 != hipSuccess) {
        printf("ERROR: Memory allocation error (hipMallocArray)\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        return false; 
    }

    // Copy reference points from host to device
    err0 = hipMemcpyToArray(ref_array_dev, 0, 0, ref, ref_nb * size_of_float * dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        return false; 
    }

    // Resource descriptor
    struct hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType         = hipResourceTypeArray;
    res_desc.res.array.array = ref_array_dev;

    // Texture descriptor
    struct hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.addressMode[0]   = hipAddressModeClamp;
    tex_desc.addressMode[1]   = hipAddressModeClamp;
    tex_desc.filterMode       = hipFilterModePoint;
    tex_desc.readMode         = hipReadModeElementType;
    tex_desc.normalizedCoords = 0;

    // Create the texture
    hipTextureObject_t ref_tex_dev = 0;
    err0 = hipCreateTextureObject(&ref_tex_dev, &res_desc, &tex_desc, NULL);
    if (err0 != hipSuccess) {
        printf("ERROR: Unable to create the texture\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        return false; 
    }

    // Compute the squared Euclidean distances
    dim3 block0(16, 16, 1);
    dim3 grid0(query_nb / 16, ref_nb / 16, 1);
    if (query_nb % 16 != 0) grid0.x += 1;
    if (ref_nb   % 16 != 0) grid0.y += 1;
    compute_distance_texture<<<grid0, block0>>>(ref_tex_dev, ref_nb, query_dev, query_nb, query_pitch, dim, dist_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false;
    }

    // Sort the distances with their respective indexes
    dim3 block1(256, 1, 1);
    dim3 grid1(query_nb / 256, 1, 1);
    if (query_nb % 256 != 0) grid1.x += 1;
    modified_insertion_sort<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, query_nb, ref_nb, k);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false;
    }

    // Compute the square root of the k smallest distances
    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (k % 16 != 0)        grid2.y += 1;
    compute_sqrt<<<grid2, block2>>>(dist_dev, query_nb, query_pitch, k);    
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false;
    }

    // Copy k smallest distances / indexes from the device to the host
    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false; 
    }

    // Memory clean-up
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev);
    hipFreeArray(ref_array_dev);
    hipDestroyTextureObject(ref_tex_dev);

    return true;
}

bool psdr_cuda::knn_cublas(const float * ref,
                int           ref_nb,
                const float * query,
                int           query_nb,
                int           dim, 
                int           k, 
                float *       knn_dist,
                int *         knn_index) {

    // Constants
    const unsigned int size_of_float = sizeof(float);
    const unsigned int size_of_int   = sizeof(int);

    // Return variables
    hipError_t  err0, err1, err2, err3, err4, err5;

    // Check that we have at least one CUDA device 
    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    // Initialize CUBLAS
    cublasInit();

    // Allocate global memory
    float * ref_dev        = NULL;
    float * query_dev      = NULL;
    float * dist_dev       = NULL;
    int   * index_dev      = NULL;
    float * ref_norm_dev   = NULL;
    float * query_norm_dev = NULL;
    size_t  ref_pitch_in_bytes;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&ref_dev,   &ref_pitch_in_bytes,   ref_nb   * size_of_float, dim);
    err1 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err2 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err3 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    err4 = hipMalloc((void**)&ref_norm_dev,   ref_nb   * size_of_float);
    err5 = hipMalloc((void**)&query_norm_dev, query_nb * size_of_float);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess || err5 != hipSuccess) {
        printf("ERROR: Memory allocation error\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Deduce pitch values
    size_t ref_pitch   = ref_pitch_in_bytes   / size_of_float;
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;

    // Check pitch values
    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false; 
    }

    // Copy reference and query data from the host to the device
    err0 = hipMemcpy2D(ref_dev,   ref_pitch_in_bytes,   ref,   ref_nb * size_of_float,   ref_nb * size_of_float,   dim, hipMemcpyHostToDevice);
    err1 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false; 
    }

    // Compute the squared norm of the reference points
    dim3 block0(256, 1, 1);
    dim3 grid0(ref_nb / 256, 1, 1);
    if (ref_nb % 256 != 0) grid0.x += 1;
    compute_squared_norm<<<grid0, block0>>>(ref_dev, ref_nb, ref_pitch, dim, ref_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Compute the squared norm of the query points
    dim3 block1(256, 1, 1);
    dim3 grid1(query_nb / 256, 1, 1);
    if (query_nb % 256 != 0) grid1.x += 1;
    compute_squared_norm<<<grid1, block1>>>(query_dev, query_nb, query_pitch, dim, query_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Computation of query*transpose(reference)
    hipblasSgemm('n', 't', (int)query_pitch, (int)ref_pitch, dim, (float)-2.0, query_dev, query_pitch, ref_dev, ref_pitch, (float)0.0, dist_dev, query_pitch);
    if (cublasGetError() != HIPBLAS_STATUS_SUCCESS) {
        printf("ERROR: Unable to execute hipblasSgemm\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;       
    }

    // Add reference points norm
    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, ref_nb / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (ref_nb   % 16 != 0) grid2.y += 1;
    add_reference_points_norm<<<grid2, block2>>>(dist_dev, query_nb, dist_pitch, ref_nb, ref_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Sort each column
    modified_insertion_sort<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, query_nb, ref_nb, k);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Add query norm and compute the square root of the of the k first elements
    dim3 block3(16, 16, 1);
    dim3 grid3(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid3.x += 1;
    if (k        % 16 != 0) grid3.y += 1;
    add_query_points_norm_and_sqrt<<<grid3, block3>>>(dist_dev, query_nb, dist_pitch, k, query_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }

    // Copy k smallest distances / indexes from the device to the host
    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false; 
    }

    // Memory clean-up and CUBLAS shutdown
    hipFree(ref_dev);
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev);
    hipFree(ref_norm_dev);
    hipFree(query_norm_dev);
    cublasShutdown();

    return true;
}

